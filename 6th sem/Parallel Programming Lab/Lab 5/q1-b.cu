#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}
int main() {
    int N;
    printf("Enter Size : ");
    scanf("%d", &N);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);
    for (int i = 0; i < N; i++) {
        A[i] = i * 1.0f;
        B[i] = i * 2.0f;
    }
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("[ ");
    for (int i = 0; i < (N < 10 ? N : 10); i++) {
        printf("%f ", C[i]);
    }
    printf(" ]\n");
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}