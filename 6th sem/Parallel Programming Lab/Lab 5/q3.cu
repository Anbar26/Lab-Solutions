#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void computeSine(float *angles, float *sines, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        sines[idx] = sinf(angles[idx]); 
    }
}

int main() {
    int N;

    printf("Enter the number of angles: ");
    scanf("%d", &N);

    int size = N * sizeof(float);

    float *h_angles = (float*)malloc(size);
    float *h_sines = (float*)malloc(size);

    printf("Enter the angles in radians:\n");
    for (int i = 0; i < N; i++) {
        printf("Angle %d: ", i + 1);
        scanf("%f", &h_angles[i]);
    }

    printf("\nAngles in Radians:\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_angles[i]);
    }
    printf("\n");

    float *d_angles, *d_sines;
    hipMalloc((void**)&d_angles, size);
    hipMalloc((void**)&d_sines, size);

    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    computeSine<<<numBlocks, blockSize>>>(d_angles, d_sines, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(h_sines, d_sines, size, hipMemcpyDeviceToHost);

    printf("\nSine of Angles:\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_sines[i]);
    }
    printf("\n");

    hipFree(d_angles);
    hipFree(d_sines);
    free(h_angles);
    free(h_sines);

    return 0;
}
