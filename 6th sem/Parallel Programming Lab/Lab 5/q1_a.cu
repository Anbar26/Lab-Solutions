#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int i = threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}
int main() {
    int N;
    printf("Enter Size: ");
    scanf("%d", &N);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);
    printf("Input Vector A:\n");
    for (int i = 0; i < N; i++) {
        scanf("%f", &A[i]);
    }
    printf("Input Vector B:\n");
    for (int i = 0; i < N; i++) {
        scanf("%f", &B[i]);
    }
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    vectorAdd<<<1, N>>>(d_A, d_B, d_C, N);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("Resultant Vector -->\n[ ");
    for (int i = 0; i < N; i++) {
        printf("%f ", C[i]);
    }
    printf("]\n");
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
