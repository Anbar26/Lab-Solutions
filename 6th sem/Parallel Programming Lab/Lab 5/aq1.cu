#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
__global__ void saxpy(float *x, float *y, float a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}
int main() {
    int N;
    printf("Enter Size: ");
    scanf("%d", &N);
    float *x, *y, a;
    float *d_x, *d_y;
    size_t size = N * sizeof(float);
    x = (float*)malloc(size);
    y = (float*)malloc(size);
    printf("Enter Scalar a: ");
    scanf("%f", &a);
    printf("Input Vector x :");
    for (int i = 0; i < N; i++) scanf("%f", &x[i]);
    printf("Input Vector y :");
    for (int i = 0; i < N; i++) scanf("%f", &y[i]);
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    saxpy<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_x, d_y, a, N);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    printf("Resultant Vector y --->\n[ ");
    for (int i = 0; i < (N < 10 ? N : 10); i++) printf("%f ", y[i]);
    printf("]\n");
    free(x);
    free(y);
    hipFree(d_x);
    hipFree(d_y);
    return 0;
}