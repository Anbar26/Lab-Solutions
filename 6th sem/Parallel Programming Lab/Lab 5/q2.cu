#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}
int main() {
    int n;
    printf("Enter the number of elements in the vectors: ");
    scanf("%d", &n);
    int *h_a = (int*)malloc(n * sizeof(int));
    int *h_b = (int*)malloc(n * sizeof(int));
    int *h_c = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    printf("Initial vector A: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    printf("Initial vector B: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_b[i]);
    }
    printf("\n");
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));
    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);
    int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vectorAdd<<<numBlocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resulting vector after addition: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_c[i]);
    }
    printf("\n");
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
