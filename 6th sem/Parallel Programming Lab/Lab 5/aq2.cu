#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
__global__ void selectionSortRows(float *matrix, int cols, int rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        for (int i = 0; i < cols - 1; i++) {
            int minIdx = i;
            for (int j = i + 1; j < cols; j++) {
                if (matrix[row * cols + j] < matrix[row * cols + minIdx]) {
                    minIdx = j;
                }
            }
            float temp = matrix[row * cols + i];
            matrix[row * cols + i] = matrix[row * cols + minIdx];
            matrix[row * cols + minIdx] = temp;
        }
    }
}
int main() {
    int rows, cols;
    printf("Enter Dimensions : ");
    scanf("%d %d", &rows, &cols);

    float *matrix, *d_matrix;
    size_t size = rows * cols * sizeof(float);
    matrix = (float*)malloc(size);

    printf("Enter Elements :\n");
    for (int i = 0; i < rows * cols; i++) scanf("%f", &matrix[i]);

    hipMalloc((void**)&d_matrix, size);
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    int blocksPerGrid = (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    selectionSortRows<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_matrix, cols, rows);

    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    printf("Sorted Matrix --->\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    free(matrix);
    hipFree(d_matrix);
    return 0;
}