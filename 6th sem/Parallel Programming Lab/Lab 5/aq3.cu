#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
__global__ void oddEvenSort(float *arr, int n, int phase) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n / 2) {
        int index = (phase % 2 == 0) ? (2 * i) : (2 * i + 1);
        if (index + 1 < n && arr[index] > arr[index + 1]) {
            float temp = arr[index];
            arr[index] = arr[index + 1];
            arr[index + 1] = temp;
        }
    }
}
int main() {
    int N;
    printf("Enter Size : ");
    scanf("%d", &N);
    float *arr, *d_arr;
    size_t size = N * sizeof(float);
    arr = (float*)malloc(size);
    printf("Enter Elements : ");
    for (int i = 0; i < N; i++) scanf("%f", &arr[i]);
    hipMalloc((void**)&d_arr, size);
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    for (int phase = 0; phase < N; phase++) {
        oddEvenSort<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_arr, N, phase);
    }
    hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
    printf("Sorted Array --->\n[ ");
    for (int i = 0; i < (N < 10 ? N : 10); i++) printf("%f ", arr[i]);
    printf("]\n");
    free(arr);
    hipFree(d_arr);
    return 0;
}