#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void ones_complement(int *arr, int *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = ~arr[idx];
    }
}

int binary_to_decimal(const char *binary) {
    int decimal = 0;
    for (int i = 0; binary[i] != '\0'; i++) {
        decimal = (decimal << 1) | (binary[i] - '0');
    }
    return decimal;
}

void decimal_to_binary(int decimal, char *binary) {
    for (int i = 31; i >= 0; i--) {
        binary[31 - i] = (decimal & (1 << i)) ? '1' : '0';
    }
    binary[32] = '\0';
}

int main() {
    int n;

    printf("Enter the number of binary numbers: ");
    scanf("%d", &n);

    char **arr = (char **)malloc(n * sizeof(char *));
    for (int i = 0; i < n; ++i) {
        arr[i] = (char *)malloc(33 * sizeof(char));
    }

    printf("Enter the binary numbers (as strings, e.g., 1010):\n");
    for (int i = 0; i < n; ++i) {
        scanf("%s", arr[i]);
    }

    int *decimal_arr = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        decimal_arr[i] = binary_to_decimal(arr[i]);
    }

    int *result = (int *)malloc(n * sizeof(int));
    int *d_arr, *d_result;

    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMalloc((void**)&d_result, n * sizeof(int));

    hipMemcpy(d_arr, decimal_arr, n * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    ones_complement<<<grid_size, block_size>>>(d_arr, d_result, n);

    hipDeviceSynchronize();

    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Original binary numbers and their ones' complements (in binary):\n");
    for (int i = 0; i < n; ++i) {
        char original_binary[33], complement_binary[33];
        decimal_to_binary(decimal_arr[i], original_binary);
        decimal_to_binary(result[i], complement_binary);
        printf("Original: %s -> One's complement: %s\n", original_binary, complement_binary);
    }

    for (int i = 0; i < n; ++i) {
        free(arr[i]);
    }
    free(arr);
    free(decimal_arr);
    free(result);
    hipFree(d_arr);
    hipFree(d_result);

    return 0;
}
