#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void selection_sort(int *arr, int n) {
    int tid = threadIdx.x;
    
    if (tid == 0) { 
        for (int i = 0; i < n - 1; i++) {
            int min_idx = i;
            for (int j = i + 1; j < n; j++) {
                if (arr[j] < arr[min_idx]) {
                    min_idx = j;
                }
            }
            int temp = arr[i];
            arr[i] = arr[min_idx];
            arr[min_idx] = temp;
        }
    }
}

int main() {
    int N;

    printf("Enter the number of elements in the array: ");
    scanf("%d", &N);

    int h_arr[N];

    printf("Enter %d elements:\n", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_arr[i]);
    }

    int *d_arr;
    hipMalloc(&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    selection_sort<<<1, 1>>>(d_arr, N);

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}