#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void convolution_kernel(float *N, float *M, float *P, int width, int mask_width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int half_mask_width = mask_width / 2;

    if (idx < width) {
        float sum = 0.0f;
        for (int j = 0; j < mask_width; j++) {
            int input_index = idx + j - half_mask_width;
            if (input_index >= 0 && input_index < width) {
                sum += N[input_index] * M[j];
            }
        }
        P[idx] = sum;
    }
}

int main() {
    int width, mask_width;

    printf("Enter the size of the input array (width): ");
    scanf("%d", &width);

    printf("Enter the size of the mask (mask_width): ");
    scanf("%d", &mask_width);

    float *N = (float*)malloc(width * sizeof(float));
    float *M = (float*)malloc(mask_width * sizeof(float));
    float *P = (float*)malloc(width * sizeof(float));

    printf("Enter the elements of the input array N:\n");
    for (int i = 0; i < width; ++i) {
        scanf("%f", &N[i]);
    }

    printf("Enter the elements of the mask M:\n");
    for (int i = 0; i < mask_width; ++i) {
        scanf("%f", &M[i]);
    }

    float *d_N, *d_M, *d_P;
    hipMalloc((void**)&d_N, width * sizeof(float));
    hipMalloc((void**)&d_M, mask_width * sizeof(float));
    hipMalloc((void**)&d_P, width * sizeof(float));

    hipMemcpy(d_N, N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 256; 
    int grid_size = (width + block_size - 1) / block_size; 

    convolution_kernel<<<grid_size, block_size>>>(d_N, d_M, d_P, width, mask_width);

    hipMemcpy(P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("Resultant Convolution Output P:\n");
    for (int i = 0; i < width; i++) {
        printf("%f ", P[i]);
    }
    printf("\n");

    free(N);
    free(M);
    free(P);
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}
