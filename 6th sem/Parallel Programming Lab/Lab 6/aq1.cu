#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int decimal_to_octal(int num) {
    int octal = 0, place = 1;
    while (num > 0) {
        octal += (num % 8) * place;
        num /= 8;
        place *= 10;
    }
    return octal;
}

__global__ void convert_to_octal(int *arr, int *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = decimal_to_octal(arr[idx]);
    }
}

int main() {
    int n;

    printf("Enter the number of integers: ");
    scanf("%d", &n);

    int *arr = (int *)malloc(n * sizeof(int));
    printf("Enter the integers:\n");
    for (int i = 0; i < n; ++i) {
        scanf("%d", &arr[i]);
    }

    int *result = (int *)malloc(n * sizeof(int));
    int *d_arr, *d_result;

    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMalloc((void**)&d_result, n * sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    convert_to_octal<<<grid_size, block_size>>>(d_arr, d_result, n);

    hipDeviceSynchronize();

    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Original integers and their octal equivalents:\n");
    for (int i = 0; i < n; ++i) {
        printf("Decimal: %d -> Octal: %d\n", arr[i], result[i]);
    }

    free(arr);
    free(result);
    hipFree(d_arr);
    hipFree(d_result);

    return 0;
}
