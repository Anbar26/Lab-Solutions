#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void odd_phase(int *arr, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx % 2 == 1 && idx < width - 1) {
        if (arr[idx] > arr[idx + 1]) {
            int temp = arr[idx];
            arr[idx] = arr[idx + 1];
            arr[idx + 1] = temp;
        }
    }
}

__global__ void even_phase(int *arr, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx % 2 == 0 && idx < width - 1) {
        if (arr[idx] > arr[idx + 1]) {
            int temp = arr[idx];
            arr[idx] = arr[idx + 1];
            arr[idx + 1] = temp;
        }
    }
}

void odd_even_transposition_sort(int *arr, int width) {
    int *d_arr;
    
    hipMalloc((void**)&d_arr, width * sizeof(int));
    hipMemcpy(d_arr, arr, width * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (width + block_size - 1) / block_size;

    int sorted = 0;
    while (!sorted) {
        sorted = 1;

        odd_phase<<<grid_size, block_size>>>(d_arr, width);
        hipDeviceSynchronize();

        even_phase<<<grid_size, block_size>>>(d_arr, width);
        hipDeviceSynchronize();

        int *h_arr = (int*)malloc(width * sizeof(int));
        hipMemcpy(h_arr, d_arr, width * sizeof(int), hipMemcpyDeviceToHost);
        
        for (int i = 1; i < width; i++) {
            if (h_arr[i - 1] > h_arr[i]) {
                sorted = 0;
                break;
            }
        }
        
        free(h_arr);
    }

    hipMemcpy(arr, d_arr, width * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int width;

    printf("Enter the size of the array: ");
    scanf("%d", &width);

    int *arr = (int*)malloc(width * sizeof(int));
    printf("Enter the elements of the array:\n");
    for (int i = 0; i < width; ++i) {
        scanf("%d", &arr[i]);
    }

    printf("Original Array: ");
    for (int i = 0; i < width; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    odd_even_transposition_sort(arr, width);

    printf("Sorted Array: ");
    for (int i = 0; i < width; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    free(arr);
    return 0;
}
